#include "cuRBM.h"

__constant__ unsigned nVis;
__constant__ unsigned nHid;
__constant__ unsigned batch;
__constant__ unsigned miniBatch;
__constant__ int *data;
__constant__ weight_t *weight;
__constant__ size_t pitch_data;
__constant__ size_t pitch_weight;

void cudaErrorCheck(hipError_t error){
  if(error != hipSuccess){
	cout << "CUDA error: " << hipGetErrorString(error) << endl;
	exit(-1);
   }
}

void deviceInit(unsigned nvisible, unsigned nhidden){
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(miniBatch), &h_miniBatch, sizeof(unsigned), 0, hipMemcpyHostToDevice));

  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(nVis), &nvisible, sizeof(unsigned), 0, hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(nHid), &nhidden, sizeof(unsigned), 0, hipMemcpyHostToDevice));

  cudaErrorCheck(hipMallocPitch((void **)&d_data, &d_pitch, len * sizeof(int), h_batch));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(data), &d_data, sizeof(int *), 0, hipMemcpyHostToDevice));
  cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(pitch_data), &d_pitch, sizeof(size_t), 0, hipMemcpyHostToDevice));

  cudaErrorCheck(hipMalloc((void **)&d_data, len * sizeof(int)));
}

void batchTransfer(int *data, unsigned nCase){
  // Copy data to device coalesced
  cudaErrorCheck(hipMemcpy2D(d_data, d_pitch, data, h_pitch, width, nCase, hipMemcpyHostToDevice));
}
