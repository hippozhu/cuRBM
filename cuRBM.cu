#include "cuRBM.h"

__constant__ unsigned nVis;
__constant__ unsigned nHid;
__constant__ unsigned batch;
__constant__ unsigned miniBatch;
__constant__ int *data;
__constant__ weight_t *weight;
__constant__ weight_t *a;
__constant__ weight_t *b;
__constant__ size_t pitch_data;
__constant__ size_t pitch_weight;

int *d_data;
weight_t *d_weight, *d_a, *d_b;
size_t d_pitch_weight, d_pitch_data;
/*
void HANDLE_ERROR(hipError_t error){
  if(error != hipSuccess){
	cout << "CUDA error: " << hipGetErrorString(error) << endl;
	exit(-1);
   }
}
*/
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}


void batchTransfer(unsigned start, unsigned nCase){
  // Copy data to device coalesced
  int *data = h_data + len * start;
  HANDLE_ERROR(hipMemcpy2D(d_data, d_pitch_data, data, h_pitch_data, width, nCase, hipMemcpyHostToDevice));
}

void runRBM(){
  for(unsigned i = 0; i < nInst; i += h_miniBatch){
    unsigned currentBatch = h_miniBatch > (nInst - i)? (nInst - i): h_miniBatch;
    batchTransfer(i, currentBatch);
    /*
    unsigned *d = (unsigned *)malloc(len * nInst * sizeof(unsigned));
    HANDLE_ERROR(hipMemcpy2D(d, h_pitch_data, d_data, d_pitch_data, 
                                width, currentBatch, hipMemcpyDeviceToHost));
    cout << *(h_data + i * len) << endl;
    cout << d[0] << endl;
    */
  }
}

void deviceInit(){
  // basic parameters to constant memory
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(miniBatch), &h_miniBatch, sizeof(unsigned), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nVis), &nvisible, sizeof(unsigned), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nHid), &nhidden, sizeof(unsigned), 0, hipMemcpyHostToDevice));

  // allocate global memory for data of mini batch 
  HANDLE_ERROR(hipMallocPitch((void **)&d_data, &d_pitch_data, len * sizeof(int), h_miniBatch));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(data), &d_data, sizeof(int *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(pitch_data), &d_pitch_data, sizeof(size_t), 0, hipMemcpyHostToDevice));

  // weights to global memory
  HANDLE_ERROR(hipMallocPitch((void **)&d_weight, &d_pitch_weight, nhidden * sizeof(weight_t), nvisible));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(weight), &d_weight, sizeof(weight_t *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(pitch_weight), &d_pitch_weight, sizeof(size_t), 0, hipMemcpyHostToDevice));
  
  // bias to global memory
  HANDLE_ERROR(hipMalloc((void **)&d_a, nvisible * sizeof(weight_t)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(a), &d_a, sizeof(weight_t *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_b, nhidden * sizeof(weight_t)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(b), &d_b, sizeof(weight_t *), 0, hipMemcpyHostToDevice));
}
