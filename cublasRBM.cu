#include "cuRBM.h"
//#include "cuRBM.cuh"

extern float *d_weight, *d_a, *d_b;

void cublasRunRBM(){
  // data
  float *m_data = (float *)malloc(sizeof(float)*ninst*nvisible);
  arrayToMatrix(m_data);
  
        hipEvent_t start, stop;
        HANDLE_ERROR(hipEventCreate(&start));
        HANDLE_ERROR(hipEventCreate(&stop));
        HANDLE_ERROR(hipEventRecord(start, NULL));
	
  float *d_data_a, *d_data_c;
  // allocate mini batch on device
  HANDLE_ERROR(hipMalloc((void **)&d_data_a, h_miniBatch * nvisible * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&d_data_c, h_miniBatch * nhidden * sizeof(float)));
  
  // weights 
  HANDLE_ERROR(hipMalloc((void **)&d_weight, nhidden * nvisible * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_weight, h_weight, nhidden * nvisible * sizeof(float), hipMemcpyHostToDevice));
  
  /*
  // bias to global memory
  HANDLE_ERROR(hipMalloc((void **)&d_a, nvisible * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_a, h_a, nvisible * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(a), &d_a, sizeof(float *), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&d_b, nhidden * sizeof(float)));
  HANDLE_ERROR(hipMemcpy(d_b, h_b, nhidden * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(b), &d_b, sizeof(float *), 0, hipMemcpyHostToDevice));
  */
  
  hipblasHandle_t handle;
  hipblasStatus_t ret;
  ret = hipblasCreate(&handle);
  CUBLAS_HANDLE_ERROR(ret);
  const float alpha = 1.0f;
  const float beta  = 0.0f;
  
  float *h_data_c = (float *)malloc(sizeof(float)*h_miniBatch*nhidden);

  for(unsigned i = 0; i < ninst; i += h_miniBatch){
    unsigned currentBatch = h_miniBatch > (ninst - i)? (ninst - i): h_miniBatch;
    HANDLE_ERROR(hipMemcpy(d_data_a, m_data + i * nvisible, currentBatch * nvisible * sizeof(float), hipMemcpyHostToDevice));
    ret = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
                      currentBatch, nhidden, nvisible, &alpha,
                      d_data_a, nvisible, d_weight, nhidden, &beta, d_data_c, h_miniBatch);
    CUBLAS_HANDLE_ERROR(ret);
    HANDLE_ERROR(hipMemcpy(h_data_c, d_data_c, sizeof(float)*nhidden*h_miniBatch, hipMemcpyDeviceToHost));
    //printArray(h_data_c, nhidden, h_miniBatch);
    cout << "result:" << h_data_c[0] << " " << h_data_c[h_miniBatch] << " " << h_data_c[1];
  }
  hipblasDestroy(handle);

        HANDLE_ERROR(hipEventRecord(stop, NULL));
        HANDLE_ERROR(hipEventSynchronize(stop));
        float msecTotal = 0.0f;
        HANDLE_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
	printf("\tcublas: %.2f msec\n", msecTotal);

  HANDLE_ERROR(hipFree(d_data_a));
  HANDLE_ERROR(hipFree(d_data_c));
  HANDLE_ERROR(hipFree(d_a));
  HANDLE_ERROR(hipFree(d_b));
  free(h_data_c);
  free(m_data);
}
